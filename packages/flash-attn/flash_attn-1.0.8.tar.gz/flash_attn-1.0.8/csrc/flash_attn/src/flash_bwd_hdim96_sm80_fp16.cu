// Copyright (c) 2023, Tri Dao.

// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_bwd_launch_template.h"

template<>
void run_mha_bwd_<96>(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    using elem_type = cutlass::half_t;
    // run_flash_bwd_loop<Flash_bwd_kernel_traits<96, 64, 128, 8, 2, 4, 4, true, elem_type>>(params, stream, configure);
    // This is very slightly faster
    run_flash_bwd_loop<Flash_bwd_kernel_traits<96, 64, 128, 8, 2, 4, 4, false, elem_type>>(params, stream, configure);
}