#include "hip/hip_runtime.h"
#define MAX_DIMENSIONS 3 // Set the number of dimensions accordingly to your array

__global__ void unroll(
    bool *arr,
    unsigned int *shape,
    unsigned long long gpu_arr_size,
    unsigned long long shape_total,
    unsigned long long dimensions_count,
    unsigned long long step,
    unsigned char order,
    unsigned long long batch_start
)
{
    unsigned long long idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long idx_full;
    unsigned int i = 0;
    unsigned int indices[MAX_DIMENSIONS];
    unsigned long long tmp;
    
    idx_full = i * step + idx;

    while (idx_full < shape_total && idx_full < gpu_arr_size)
    {
        tmp = idx_full + batch_start; // add batch_start to account for the offset
        // Compute the indices
        for (unsigned int j = 0; j < dimensions_count; ++j)
        {
            unsigned int dimension = (order == 0) ? dimensions_count - j - 1 : j;
            // Modulo by the dimension size
            indices[dimension] = tmp % shape[dimension];
            // Divide by the dimension size
            tmp /= shape[dimension];
        }
        
        for (unsigned int j = 0; j < dimensions_count; ++j)
        {
            // j is the dimension

            // Your code there ++
            if (indices[j] == 3)
            {
                // Set true if any index equals to 3
                arr[idx_full] = true;
                break;
            }
            // Your code there --            
            
        }
        i += 1;
        idx_full = i * step + idx;
    }
    // Free the memory
    delete[] indices;
}
